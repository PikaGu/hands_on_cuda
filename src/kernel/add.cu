#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "kernel.h"

__global__ void add(const float* a, const float* b, float* c, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

__global__ void add_float2(const float* a, const float* b, float* c, int n) {
  int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
  if (idx < n) {
    
  }
}

__global__ void add_float4(const float* a, const float* b, float* c, int n) {
  int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
  if (idx < n) {
    
  }
}

void launch_vector_add(const float* A, const float* B, float* C, int N, int version=0) {
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, N * sizeof(float));
  hipMalloc((void**)&d_B, N * sizeof(float));
  hipMalloc((void**)&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (N + threads - 1) / threads;
  switch (version) {
  case 1:
    add_float2<<<blocks, threads>>>(d_A, d_B, d_C, N);
    break;
  case 2:
    add_float4<<<blocks, threads>>>(d_A, d_B, d_C, N);
    break;
  default:
    add<<<blocks, threads>>>(d_A, d_B, d_C, N);
    break;
  }

  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
