#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "kernel.h"

__global__ void add(const float* a, const float* b, float* c, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

void launch_vector_add(const float* A, const float* B, float* C, int N) {
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, N * sizeof(float));
  hipMalloc((void**)&d_B, N * sizeof(float));
  hipMalloc((void**)&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (N + threads - 1) / threads;
  add<<<blocks, threads>>>(d_A, d_B, d_C, N);

  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
